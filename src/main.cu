#include "hip/hip_runtime.h"

#include <cstddef>  // std::size_t type
#include <iostream> // std::cout, std::endl
#include <vector>

#include "algorithm_choices.h"
#include "data_generator.h"
#include "data_types.h"
<<<<<<< Updated upstream
#include "cuda_common.h"
=======


template <typename T>
void print_matrix(T* matrix, int m, int n) {
    for (int i = 0; i < m; i++) {
        std::cout << "| ";
        for (int j = 0; j < n; j++) {
            std::cout << matrix[i * m + j] << " ";

        }
        std::cout << "|" << std::endl;
    }
}
/**
 * Runs timing tests on a CUDA graph implementation.
 * Consists of independently constructing the graph and then
 * modifying it to its two-hop neighbourhood.
 */
template < typename DeviceGraph >
void run(DeviceGraph g, csc485b::a2::edge_t const* d_edges, std::size_t m)
{
    
    hipDeviceSynchronize();
    auto const build_start = std::chrono::high_resolution_clock::now();

    // this code doesn't work yet!
    csc485b::a2::gpu::build_graph << < 1, 1 >> > (g, d_edges, m);

    hipDeviceSynchronize();
    auto const reachability_start = std::chrono::high_resolution_clock::now();


    // neither does this!   
    unsigned int tiling_size = 2;
    unsigned int matrix_size = sqrt(g.n);
    unsigned int num_block = matrix_size / tiling_size;
    csc485b::a2::gpu::two_hop_reachability <<< {num_block, num_block}, { tiling_size, tiling_size } >>> (g);

    
    hipDeviceSynchronize();
    auto const end = std::chrono::high_resolution_clock::now();

    
    std::cout << "Build time: "
        << std::chrono::duration_cast<std::chrono::microseconds>(reachability_start - build_start).count()
        << " us"
        << std::endl;
    
    std::cout << "Reachability time: "
        << std::chrono::duration_cast<std::chrono::microseconds>(end - reachability_start).count()
        << " us"
        << std::endl;
    
}

/**
 * Allocates space for a dense graph and then runs the test code on it.
 */
void run_dense(csc485b::a2::edge_t const* d_edges, std::size_t n, std::size_t m)
{
    using namespace csc485b;

    // allocate device DenseGraph
    a2::node_t* d_matrix;
    hipMalloc((void**)&d_matrix, sizeof(a2::node_t) * n * n);
    a2::DenseGraph d_dg{ n, d_matrix };

    run(d_dg, d_edges, m);

    // check output?
    std::vector< a2::node_t > host_matrix(d_dg.matrix_size());
    a2::DenseGraph dg{ n, host_matrix.data() };
    hipMemcpy(dg.adjacencyMatrix, d_dg.adjacencyMatrix, sizeof(a2::node_t) * d_dg.matrix_size(), hipMemcpyDeviceToHost);
    std::copy(host_matrix.cbegin(), host_matrix.cend(), std::ostream_iterator< a2::node_t >(std::cout, " "));

    // clean up
    hipFree(d_matrix);
}

/**
 * Allocates space for a sparse graph and then runs the test code on it.
 */
void run_sparse(csc485b::a2::edge_t const* d_edges, std::size_t n, std::size_t m)
{
    using namespace csc485b;

    // allocate device SparseGraph
    a2::node_t* d_offsets, * d_neighbours;
    hipMalloc((void**)&d_offsets, sizeof(a2::node_t) * n);
    hipMalloc((void**)&d_neighbours, sizeof(a2::node_t) * m);
    a2::SparseGraph d_sg{ n, m, d_offsets, d_neighbours };

    run(d_sg, d_edges, m);

    // clean up
    hipFree(d_neighbours);
    hipFree(d_offsets);
}
>>>>>>> Stashed changes

int main()
{
    std::size_t const n = 32*32;
    std::size_t const switch_at = 3 * (n >> 2);

<<<<<<< Updated upstream
    auto data = csc485b::a1::generate_uniform< element_t >(n);
    csc485b::a1::cpu::run_cpu_baseline(data, switch_at, n);
    csc485b::a1::gpu::run_gpu_soln(data, switch_at, n);
=======
    // Create input
    std::size_t constexpr n = 4;
    std::size_t constexpr expected_degree = n >> 1;

    a2::edge_list_t const graph = a2::generate_graph(n, n * expected_degree);
    std::size_t const m = graph.size();

    // lazily echo out input graph
    for (auto const& e : graph)
    {
        std::cout << "(" << e.x << "," << e.y << ") ";
    }

    // allocate and memcpy input to device
    a2::edge_t* d_edges;
    hipMalloc((void**)&d_edges, sizeof(a2::edge_t) * m);
    hipMemcpyAsync(d_edges, graph.data(), sizeof(a2::edge_t) * m, hipMemcpyHostToDevice);

    // run your code!
    run_dense(d_edges, n, m);
   // run_sparse(d_edges, n, m);
>>>>>>> Stashed changes

    std::cout << "hello world!" << std::endl;
    return EXIT_SUCCESS;
}